
#include <hip/hip_runtime.h>
// nvcc -cubin -arch=sm_50 segment_count.cu

extern "C"
__global__ void count_pixels_per_segment(int* image, int* segment_counts, int* num_segments, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int segment = image[y * width + x];
        int prevCount = atomicAdd(&segment_counts[segment], 1);
        if (prevCount == 0)
        {
            atomicAdd(num_segments, 1);
        }
    }
}